#include "hip/hip_runtime.h"
#pragma comment (lib, "cublas.lib")

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <ctime>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <hipblas.h>
#include <hipblas.h>

using namespace std;

const int N = 1 << 14;
#define V 0.2
#define T 2

#define CUDA_CHECK_RETURN(value) ((hipError_t)value != hipSuccess) ? printf("Error %s at line %d in the file %s\n", hipGetErrorString((hipError_t)value), __LINE__, __FILE__) : printf("")

void output(float* a, int n)
{
    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            printf("%g ", a[i + j * n]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void sgemvblas(float *a, float *x, float *y, int count) 
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < count)
	{
		float res = 0;
		for(int j = 0; j < count; j++)
		{
			res += a[i * count + j] * x[j];
		}
		y[i] = res;
	}
}

struct dp
{
	float *A, *B;
  	int count;
  	dp(float *_A, float *_B, int _count): A(_A), B(_B), count(_count){};
  	__host__ __device__
  	float operator()(size_t idx)
	{
   		float sum = 0.0f;
    		int row = idx / count;
    		int col = idx - (row * count);
    		for (int i = 0; i < count; i++)
		{
      			sum += A[col * i + row] * B[i];			
		}
    		return sum;
	}
};

void sgemv(int count, thrust::device_vector<float> &data, thrust::device_vector<float> &other, thrust::device_vector<float> &result)
{
	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(count), result.begin(), dp(thrust::raw_pointer_cast(data.data()), thrust::raw_pointer_cast(other.data()), count));
}

int main()
{
	srand(time(NULL));
	
	hipEvent_t start, stop;

	float time = 0;
	float alpha = 1;
	float max[15];

	for(int i = 0; i < 15; i++)
	{
		max[i] = 0;
	}

	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));

	hipblasHandle_t cublas_handle;
	CUDA_CHECK_RETURN(hipblasCreate(&cublas_handle)); 

	int count = 1;
	int k = 0;

	for(int i = 0; i < 10; i++)
	{
		count = 1;
		while(count <= N)
		{
			float *a;
			float *x;
			float *y;

			float *a_dev;
			float *x_dev;
			float *y_dev;

			CUDA_CHECK_RETURN(hipMalloc((void **)&a_dev, count * count * sizeof(float)));	
			CUDA_CHECK_RETURN(hipMalloc((void **)&x_dev, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void **)&y_dev, count * sizeof(float)));
	
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&a, count * count * sizeof(float)));
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&x, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&y, count * sizeof(float)));

			for(int i = 0; i < count; i++)
			{
				x[i] = rand() % 1000;
				y[i] = 0;
				for(int j = 0; j < count; j++)
				{
					a[i * count + j] = rand() % 1000;	
				}
			}

    			CUDA_CHECK_RETURN(hipMemcpy(a_dev, a, count * count * sizeof(float), hipMemcpyHostToDevice));
    			CUDA_CHECK_RETURN(hipMemcpy(x_dev, x, count * sizeof(float), hipMemcpyHostToDevice));
    			CUDA_CHECK_RETURN(hipMemcpy(y_dev, y, count * sizeof(float), hipMemcpyHostToDevice));

			dim3 threadBlock(1024, 1);
			dim3 blockGrid(count / 1024 + 1, 1, 1);

       			CUDA_CHECK_RETURN(hipEventSynchronize(start));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	
			sgemvblas <<< blockGrid, threadBlock >>> (a_dev, x_dev, y_dev, count); 	
	
			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));

    			CUDA_CHECK_RETURN(hipMemcpy(y, y_dev, count * sizeof(float), hipMemcpyDeviceToHost));

			printf("%d - CUDA: %f ms\n", count, time);
			max[k] += time;

			CUDA_CHECK_RETURN(hipHostFree(a));
			CUDA_CHECK_RETURN(hipHostFree(x));
			CUDA_CHECK_RETURN(hipHostFree(y));

			CUDA_CHECK_RETURN(hipFree(a_dev));
			CUDA_CHECK_RETURN(hipFree(x_dev));
			CUDA_CHECK_RETURN(hipFree(y_dev));

			count <<= 1;
			k++;
		}
		k = 0;
	}
	
	for(int i = 0; i < 15; i++)
	{
		cout << max[i] / 10 << endl;
		max[i] = 0;
	}
	
	for(int i = 0; i < 10; i++)
	{
		count = 1;
		while(count <= N)
		{
  			thrust::device_vector<float> data(count * count, 2);
  			thrust::device_vector<float> other(count, 5);
  			thrust::device_vector<float> result(count, 0);

 			CUDA_CHECK_RETURN(hipEventSynchronize(start));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));

  			sgemv(count, data, other, result);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));
  			CUDA_CHECK_RETURN(hipDeviceSynchronize());

  			printf("%d - THRUST: %f ms\n", count, time);
			max[k] += time;
			count <<= 1;
			k++;
		}
		k = 0;
	}
	for(int i = 0; i < 15; i++)
	{
		cout << max[i] / 10 << endl;
		max[i] = 0;
	}

/////////////////////////////////////////////////////CUBLAS//////////////////////////////////////////////////////////////
	for(int i = 0; i < 10; i++)
	{
		count = 1;
		while(count <= N)
		{	
			float *a;
			float *x;
			float *y;

			float *a_dev;
			float *x_dev;
			float *y_dev;

			CUDA_CHECK_RETURN(hipMalloc((void **)&a_dev, count * count * sizeof(float)));	
			CUDA_CHECK_RETURN(hipMalloc((void **)&x_dev, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void **)&y_dev, count * sizeof(float)));

			CUDA_CHECK_RETURN(hipHostMalloc((void **)&a, count * count * sizeof(float)));
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&x, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&y, count * sizeof(float)));

			for(int i = 0; i < count; i++)
			{
				x[i] = rand() % 1000;
				y[i] = 0;
				for(int j = 0; j < count; j++)
				{
					a[i * count + j] = rand() % 1000;	
				}
			}
			CUDA_CHECK_RETURN(hipblasSetMatrix(count, count, sizeof(float), (void *)a, count, (void *)a_dev, count));
			CUDA_CHECK_RETURN(hipblasSetVector(count, sizeof(float), (void *)x, 1, (void *)x_dev, 1));
			CUDA_CHECK_RETURN(hipblasSetVector(count, sizeof(float), (void *)y, 1, (void *)y_dev, 1));

			CUDA_CHECK_RETURN(hipEventSynchronize(start));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));

			CUDA_CHECK_RETURN(hipblasSgemv(cublas_handle, HIPBLAS_OP_N, count, count, &alpha, a_dev, count, x_dev, 1, &alpha, y_dev, 1));

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));

			CUDA_CHECK_RETURN(hipblasGetMatrix(count, count, sizeof(float), (void *)a_dev, count, (void *)a, count));
			CUDA_CHECK_RETURN(hipblasGetVector(count, sizeof(float), (void *)x_dev, 1, (void *)x, 1));
			CUDA_CHECK_RETURN(hipblasGetVector(count, sizeof(float), (void *)y_dev, 1, (void *)y, 1));

			printf("%d - cuBLAS: %f ms\n", count, time);
			max[k] += time;

			CUDA_CHECK_RETURN(hipHostFree(a));
			CUDA_CHECK_RETURN(hipHostFree(x));
			CUDA_CHECK_RETURN(hipHostFree(y));

			CUDA_CHECK_RETURN(hipFree(a_dev));
			CUDA_CHECK_RETURN(hipFree(x_dev));
			CUDA_CHECK_RETURN(hipFree(y_dev));

			count <<= 1;
			k++;
        	}
		k = 0;
	}

	for(int i = 0; i < 15; i++)
	{
		cout << max[i] / 10 << endl;
		max[i] = 0;
	}

	CUDA_CHECK_RETURN(hipblasDestroy(cublas_handle));
	
	return 0;
}