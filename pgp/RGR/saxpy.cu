#include "hip/hip_runtime.h"
#pragma comment (lib, "cublas.lib")

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <ctime>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <hipblas.h>
#include <hipblas.h>

using namespace std;

const int N = 1 << 26;
#define V 0.2
#define T 2

#define CUDA_CHECK_RETURN(value) ((hipError_t)value != hipSuccess) ? printf("Error %s at line %d in the file %s\n", hipGetErrorString((hipError_t)value), __LINE__, __FILE__) : printf("")


__global__ void saxpyblas(float *x, float *y, float *z) 
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	z[i] = x[i] * (V * T) + y[i];
}

struct functor 
{
    const float koef;
    functor(float _koef) : koef(_koef) {}
    __host__ __device__ float operator()(float x, float y) 
    { 
        return koef * x + y; 
    }
};

void saxpy(float _koef, thrust::device_vector<float> &x, thrust::device_vector<float> &y)
{
    functor func(_koef);
    thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

int main()
{
	srand(time(NULL));
	
	hipEvent_t start, stop;

	float time;
        float alpha = V * T;	
	float max[27];
	int k = 0;
	for(int i = 0; i < 27; i++)
	{
		max[i] = 0;
	}

	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));

	hipblasHandle_t cublas_handle;
	CUDA_CHECK_RETURN(hipblasCreate(&cublas_handle)); 

    	hipDeviceProp_t deviceProp;
	CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProp, 0));

	int blocks = 0;
	int threads = 0;
	int count = 1;

	for(int i = 0; i < 10; i++)
	{
		count = 1;
		while(count <= N)
		{
			float *x;
			float *y;
			float *z;

			float *x1;
			float *y1;
			float *z1;

			CUDA_CHECK_RETURN(hipMalloc((void **)&x1, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void **)&y1, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void **)&z1, count * sizeof(float)));

			CUDA_CHECK_RETURN(hipHostMalloc((void **)&x, count * sizeof(float)));	
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&y, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&z, count * sizeof(float)));

			for(int i = 0; i < count; i++)
			{
				x[i] = rand() % 1000;
				y[i] = rand() % 1000;
			}

    			CUDA_CHECK_RETURN(hipMemcpy(x1, x, count * sizeof(float), hipMemcpyHostToDevice));
    			CUDA_CHECK_RETURN(hipMemcpy(y1, y, count * sizeof(float), hipMemcpyHostToDevice));
	
			if(count <= deviceProp.maxThreadsPerBlock)
			{
				threads = count;
				blocks = 1;	
			}
			else
			{
				threads = deviceProp.maxThreadsPerBlock;
				blocks = count / 1024;
			}

       			CUDA_CHECK_RETURN(hipEventSynchronize(start));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));
	
			saxpyblas <<< blocks, threads >>> (x1, y1, z1); 	

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));
	
    			CUDA_CHECK_RETURN(hipMemcpy(z, z1, count * sizeof(float), hipMemcpyDeviceToHost));

			printf("%d - CUDA: %f ms\n", count, time);
			max[k] += time;

			CUDA_CHECK_RETURN(hipHostFree(x));
			CUDA_CHECK_RETURN(hipHostFree(y));
			CUDA_CHECK_RETURN(hipHostFree(z));

			CUDA_CHECK_RETURN(hipFree(x1));
			CUDA_CHECK_RETURN(hipFree(y1));
			CUDA_CHECK_RETURN(hipFree(z1));

			count <<= 1;
			k++;
		}
		k = 0;
	}

	for(int i = 0; i < 27; i++)
	{
		cout << max[i] / 10 * 1000 << endl;
		max[i] = 0;
	}

	for(int i = 0; i < 10; i++)
	{
		count = 1;
		while(count <= N)
		{	
			thrust::host_vector<float> cpumem1(count);
			thrust::host_vector<float> cpumem2(count);

			for (int i = 0; i < count; i++) 
      	  		{
	   			cpumem1[i] = rand() % 1000;
	    			cpumem2[i] = rand() % 1000;
			}

			thrust::device_vector<float> gpumem1 = cpumem1;
			thrust::device_vector<float> gpumem2 = cpumem2;

       			CUDA_CHECK_RETURN(hipEventSynchronize(start));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));

			saxpy(V * T, gpumem2, gpumem1);

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));

			printf("%d - Thrust: %f ms\n", count, time);
			max[k] += time;

			count <<= 1;
			k++;
		}
		k = 0;
	}

	for(int i = 0; i < 27; i++)
	{
		cout << max[i] / 10 * 1000 << endl;
		max[i] = 0;
	}

	for(int i = 0; i < 10; i++)
	{
		count = 1;
		while(count <= N)
		{	
			float *host_x;
			float *host_y;

			float *dev_x;
			float *dev_y;

			CUDA_CHECK_RETURN(hipMalloc((void **)&dev_x, count * sizeof(float)));
			CUDA_CHECK_RETURN(hipMalloc((void **)&dev_y, count * sizeof(float)));
	
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&host_x, count * sizeof(float)));	
			CUDA_CHECK_RETURN(hipHostMalloc((void **)&host_y, count * sizeof(float)));	

			for (int i = 0; i < count; i++) 
       			{
            			host_x[i] = rand() % 1000;
	    			host_y[i] = rand() % 1000;
			}

			CUDA_CHECK_RETURN(hipblasSetVector(count, sizeof(float), host_x, 1, dev_x, 1));
			CUDA_CHECK_RETURN(hipblasSetVector(count, sizeof(float), host_y, 1, dev_y, 1));

			CUDA_CHECK_RETURN(hipEventSynchronize(start));
			CUDA_CHECK_RETURN(hipEventRecord(start, 0));

			CUDA_CHECK_RETURN(hipblasSaxpy(cublas_handle, count, &alpha, dev_x, 1, dev_y, 1));

			CUDA_CHECK_RETURN(hipDeviceSynchronize());
			CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
			CUDA_CHECK_RETURN(hipEventSynchronize(stop));
			CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));

			CUDA_CHECK_RETURN(hipblasGetVector(count, sizeof(float), dev_x, 1, host_x, 1));
			CUDA_CHECK_RETURN(hipblasGetVector(count, sizeof(float), dev_y, 1, host_y, 1));

			printf("%d - cuBLAS: %f ms\n", count, time);
			max[k] += time;

			CUDA_CHECK_RETURN(hipHostFree(host_x));
			CUDA_CHECK_RETURN(hipHostFree(host_y));

			CUDA_CHECK_RETURN(hipFree(dev_x));
			CUDA_CHECK_RETURN(hipFree(dev_y));

			count <<= 1;
			k++;
		}
		k = 0;
        }

	for(int i = 0; i < 27; i++)
	{
		cout << max[i] / 10 * 1000 << endl;
		max[i] = 0;
	}

	CUDA_CHECK_RETURN(hipblasDestroy(cublas_handle));
	
	return 0;
}