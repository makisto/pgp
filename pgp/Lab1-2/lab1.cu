#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sum(float* a, float* b, float* c)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x; 
    //int index = blockIdx.x;
    c[index] = a[index] + b[index];
}

#define CUDA_CHECK_RETURN(value) ((hipError_t)value != hipSuccess) ? printf("Error %s at line %d in the file %s\n", hipGetErrorString((hipError_t)value), __LINE__, __FILE__) : printf("") 

int main()
{ 
    int n, k;
    scanf("%d%d", &n, &k);

    float* a = new float[n * k];
    float* b = new float[n * k];
    float* c = new float[n * k];

    for(int i = 0; i < n * k; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    float* dev1;
    float* dev2;
    float* dev3;

    float elapsedTime;
    hipEvent_t start, stop;

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));

    CUDA_CHECK_RETURN(hipMalloc((void**)&dev1, n * k * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&dev2, n * k * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&dev3, n * k * sizeof(float)));

    CUDA_CHECK_RETURN(hipMemcpy(dev1, a, n * k * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dev2, b, n * k * sizeof(float), hipMemcpyHostToDevice));
   
    CUDA_CHECK_RETURN(hipEventRecord(start, 0));
    sum <<< n, k >>> (dev1, dev2, dev3);
    CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));

    CUDA_CHECK_RETURN(hipGetLastError());

    CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
  
    fprintf(stderr, "gTest took %g\n", elapsedTime);

    CUDA_CHECK_RETURN(hipEventDestroy(start));
    CUDA_CHECK_RETURN(hipEventDestroy(stop));

    CUDA_CHECK_RETURN(hipMemcpy(c, dev3, n * k * sizeof(float), hipMemcpyDeviceToHost));

    for(int i = (n * k) - 5; i < n * k; i++)
    {
        printf("Element #%i: %f\n", i, c[i]);
    }

    free(a);
    free(b);
    free(c);

    CUDA_CHECK_RETURN(hipFree(dev1));
    CUDA_CHECK_RETURN(hipFree(dev2));
    CUDA_CHECK_RETURN(hipFree(dev3));

    return 0;
}
