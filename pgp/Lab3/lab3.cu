#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gTest(float* a)
{
    a[threadIdx.x + blockDim.x * blockIdx.x] = (float)((threadIdx.x + blockDim.x * blockIdx.x) * 2);
}

int main()
{
    int m, n, k;
    scanf("%d%d%d", &m, &n, &k);

    float* mas = new float[m];
    float* da;

    int dev;
    hipSetDevice(dev); 
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Warp size: %d\n", deviceProp.warpSize);
    printf("Max number of warps/MP: %d\n", deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize);
    printf("Max number of blocks/MP: 8\n");
   
    printf("Occupancy: %g\n", (float)(k * 8) / (float)((deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize) * deviceProp.warpSize));

    hipMalloc((void**)&da, m * sizeof(float));
    gTest <<< n, k >>> (da);
    hipDeviceSynchronize();
    hipMemcpy(mas, da, m * sizeof(float), hipMemcpyDeviceToHost);
    
    for(int i = m - 4; i < m; i++)
    {
        printf("%g\n", mas[i]);
    }
    free(mas);
    hipFree(da);
 
    return 0;
}
