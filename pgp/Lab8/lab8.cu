#pragma comment (lib, "cublas.lib")

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <hipblas.h>
#include <hipblas.h>

using namespace std;

const int N = 1<<24;
#define V 0.2
#define T 2

#define CUDA_CHECK_RETURN(value) ((hipError_t)value != hipSuccess) ? printf("Error %s at line %d in the file %s\n", hipGetErrorString((hipError_t)value), __LINE__, __FILE__) : printf("")

struct functor 
{
    const float koef;
    functor(float _koef) : koef(_koef) {}
    __host__ __device__ float operator()(float x, float y) 
    { 
        return koef * x + y; 
    }
};

void saxpy(float _koef, thrust::device_vector<float> &x, thrust::device_vector<float> &y)
{
    functor func(_koef);
    thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

int main()
{
	float *FunctionData = new float[N];
	
	float *host_x;
	float *host_y;
	float *dev_x;
	float *dev_y;

	hipEvent_t start, stop;

	thrust::host_vector<float> cpumem1(N);
	thrust::host_vector<float> cpumem2(N);

	float time;
        float alpha = V * T;

	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));

	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_x, N * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&dev_y, N * sizeof(float)));
	
	CUDA_CHECK_RETURN(hipHostMalloc((void **)&host_x, N * sizeof(float)));	
	CUDA_CHECK_RETURN(hipHostMalloc((void **)&host_y, N * sizeof(float)));

	hipblasHandle_t cublas_handle;
	CUDA_CHECK_RETURN(hipblasCreate(&cublas_handle)); 

	for (int i = 0; i < N; i++) 
        {
	    FunctionData[i] = rand() % 1000;
	    cpumem1[i] = FunctionData[i];
	    cpumem2[i] = FunctionData[i];
            host_x[i] = FunctionData[i];
	    host_y[i] = FunctionData[i];
	}
        cout << endl;

	thrust::device_vector<float> gpumem1 = cpumem1;
	thrust::device_vector<float> gpumem2 = cpumem2;

        CUDA_CHECK_RETURN(hipEventSynchronize(start));
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

	saxpy(V * T, gpumem2, gpumem1);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));

	printf("Thrust: %f ms\n", time);
	/*for (int i = 0; i < N; i++)
        {
	    cout << gpumem1[i] << " ";
        }
	cout << endl;*/

	CUDA_CHECK_RETURN(hipblasSetVector(N, sizeof(float), host_x, 1, dev_x, 1));
	CUDA_CHECK_RETURN(hipblasSetVector(N, sizeof(float), host_y, 1, dev_y, 1));

	CUDA_CHECK_RETURN(hipEventSynchronize(start));
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

	CUDA_CHECK_RETURN(hipblasSaxpy(cublas_handle, N, &alpha, dev_x, 1, dev_y, 1));

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&time, start, stop));

	CUDA_CHECK_RETURN(hipblasGetVector(N, sizeof(float), dev_x, 1, host_x, 1));
	CUDA_CHECK_RETURN(hipblasGetVector(N, sizeof(float), dev_y, 1, host_y, 1));

	printf("cuBLAS: %f ms\n", time);
	/*for (int i = 0; i < N; i++)
        {
	    cout << host_y[i] << " ";
	}*/
       
	CUDA_CHECK_RETURN(hipHostFree(host_x));
	CUDA_CHECK_RETURN(hipHostFree(host_y));
	CUDA_CHECK_RETURN(hipFree(dev_x));
	CUDA_CHECK_RETURN(hipFree(dev_y));
	CUDA_CHECK_RETURN(hipblasDestroy(cublas_handle));
	
	return 0;
}